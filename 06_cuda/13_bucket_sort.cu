
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void bucketSort(int range, int bucket[], int key[], int offset[])
{
  int i = threadIdx.x;
  for (int j = 1; j<range; j<<=1) {
    offset[i] = bucket[i];
    if(i>=j) bucket[i] += offset[i-j];
  }

  for (int j=0; bucket[i]>0; bucket[i]--)
    key[j++] = i;
}

int main() 
{
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key, n*sizeof(int));

  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }

  printf("\n");

  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
  int *offset;
  hipMallocManaged(&offset, range*sizeof(int)); 
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }
  bucketSort<<<1, n>>>(range, bucket, key, offset);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(key);
  hipFree(bucket);
}
